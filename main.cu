
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <string>
#include <fstream>
#include <sstream>

void read_pgm_file(std::string filename, int* image, int* new_image);
void write_pgm_file(std::string filename, int* new_image);
bool isNumber(char number[]);

__global__
void filter(int* image, int* new_image)
{
    int gaussian_kernel[9][9] = {
        {0, 0, 3,   2,   2,   2, 3, 0, 0},
        {0, 2, 3,   5,   5,   5, 3, 2, 0},
        {3, 3, 5,   3,   0,   3, 5, 3, 3},
        {2, 5, 3, -12, -23, -12, 3, 5, 2},
        {2, 5, 0, -23, -40, -23, 0, 5, 2},
        {2, 5, 3, -12, -23, -12, 3, 5, 2},
        {3, 3, 5,   3,   0,   3, 5, 3, 3},
        {0, 2, 3,   5,   5,   5, 3, 2, 0},
        {0, 0, 3,   2,   2,   2, 3, 0, 0},
    };

    size_t max_pixels = 65536 / (blockDim.x * gridDim.x);
    size_t index = (blockIdx.x * blockDim.x + threadIdx.x) * max_pixels;

    for (int i = index; i < index + max_pixels && index < 65536; i++)
    {
        // Convert 1D array of pixel to get 2D coordinate.
        size_t pixel_x = i % 256;
        size_t pixel_y = i / 256;
        int new_pixel_val = 0;

        for (size_t row = 0; row < 9; row++)
        {
            for (size_t col = 0; col < 9; col++)
            {
                // Subtract 4 to center the gaussian filter on the pixel.
                int mapped_gauss_x = pixel_x + col - 4;
                int mapped_gauss_y = pixel_y + row - 4; 

                size_t gauss_idx = mapped_gauss_x + 256 * mapped_gauss_y;

                if (mapped_gauss_x >= 0 && mapped_gauss_x <= 255 && mapped_gauss_y >= 0 && mapped_gauss_y <= 255)
                {
                    new_pixel_val += gaussian_kernel[col][row] * image[gauss_idx];
                }
            }
        }

        if (new_pixel_val > 255)
        {
            new_pixel_val = 255;
        }

        if (new_pixel_val < 0)
        {
            new_pixel_val = 0;
        }
        
        new_image[i] = new_pixel_val;
    }
}

int main(int argc, char *argv[])
{
    const size_t N = 256 * 256; // 1M elements
    int *image;
    int *new_image;

    if (argc != 3 || !isNumber(argv[1]) || !isNumber(argv[2]) || 65536 % (atoi(argv[1]) * atoi(argv[2])) != 0)
    {
        std::cout << "Usage:\n\n./main.exe <#Blocks> <#Threads>\n\nRequired: 65536 % (#Blocks * #Threads) == 0\n" << std::endl;
        exit(0);
    }

    int number_of_blocks = atoi(argv[1]);
    int number_of_threads = atoi(argv[2]);

    hipMallocManaged(&image, N * sizeof(int));
    hipMallocManaged(&new_image, N * sizeof(int));

    read_pgm_file("pepper.ascii.pgm", image, new_image);

    // Performance varies depending on total # threads.
    // Try running nvprof ./main.exe using different combinations
    // of blocks and threads to see the performance difference.
    filter<<<number_of_blocks, number_of_threads>>>(image, new_image);

    hipDeviceSynchronize();
    write_pgm_file("output.pgm", new_image);

    // Free memory
    hipFree(image);
    hipFree(new_image);

    return 0;
}

void read_pgm_file(std::string filename, int* image, int* new_image)
{
    std::ifstream file;
    file.open(filename);

    int line_count = 0;
    int index = 0;
    std::string line;
    while (std::getline(file, line))
    {
        // Read every line past the first 4 lines.
        if (line_count > 3)
        {
            int pixel_val;
            std::stringstream line_stream(line);

            // Read in integer values of PGM file to image array.
            while (line_stream >> pixel_val)
            {
                image[index] = pixel_val;
                new_image[index] = 0;
                index++;
            }
        }
        else
        {
            line_count++;
        }
    }

    file.close();
}

void write_pgm_file(std::string filename, int* new_image)
{
    std::ofstream output_file;
    output_file.open(filename);

    // Start with header info.
    std::string new_file = "P2 \n256 256 \n255\n";

    // Write 17 pixel values per line.
    for (size_t i = 0; i < 65536; i++)
    {
        if (i % 17 == 0 && i != 0)
        {
            new_file += "\n";
        }

        new_file += std::to_string(new_image[i]);

        if ((i + 1) % 17 != 0)
        {
            new_file += " ";
        }
    }

    new_file += "\n";

    output_file << new_file;
    output_file.close();
}

// Used to check input arguments are numbers.
bool isNumber(char number[])
{
    int i = 0;

    //checking for negative numbers
    if (number[0] == '-')
        i = 1;
    for (; number[i] != 0; i++)
    {
        //if (number[i] > '9' || number[i] < '0')
        if (!isdigit(number[i]))
            return false;
    }
    return true;
}
